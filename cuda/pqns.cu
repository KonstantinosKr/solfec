#include "hip/hip_runtime.h"
/*
 * pqns.cu
 * Copyright (C) 2010, Tomasz Koziara (t.koziara AT gmail.com)
 * --------------------------------------------------------------
 * projected quasi-Newton solver
 */

/* This file is part of Solfec.
 * Solfec is free software: you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your
 * option) any later version.
 *
 * Solfec is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public
 * License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with Solfec. If not, see <http://www.gnu.org/licenses/>. */

#include <iostream>
#include "alg.h"
using std::cerr;
using std::endl;

/* compressed row block matrix - vector product */
__global__ void U_WR_B (const int num, const int *ptr, const int *adj,
  const float *W0, const float *R0, const float *B0, float *U0)
{
  int con = blockDim.x * blockIdx.x + threadIdx.x;

  if (con < num)
  {
    const float *W = W0;
    const float *B = &B0[con*3];
    float WR[3] = {B[0], B[1], B[2]};
    int j, j0 = ptr [con], j1 = ptr [con+1];
    for (j = j0; j < j1; j ++, W += 9)
    { 
      const float *R = &R0[adj[j]*3];
      NVADDMUL (WR, W, R, WR);
    }
    float *U = &U0[con*3];
    COPY (WR, U);
  }
}

/* constraint kinds */
enum {_CONTACT_ = 0, _FIXPNT_, _FIXDIR_, _VELODIR_, _RIGLNK_, _GLUE_};

/* diagonal solver */
__global__ void solve (const int dynamic, const float step, const float theta, const float eps,
  const int num, const int *kind, const float *mat0, const int *ptr, const float *W0,
  const float *V0, const float *U0, float *DR0, float *R0)
{
  int con = blockDim.x * blockIdx.x + threadIdx.x;

  const float *U = &U0[con*3],
              *V = &V0[con*3],
              *mat = &mat0[con*4],
              *W = &W0[ptr[con]*9];

  float *R = &R0[con*3],
        *DR = &DR0[con*3];

  float x [3], T [9], gamma = 1.0 - theta;

  switch (kind [con])
  {
  case _FIXPNT_:
  case _GLUE_:
  {
    if (dynamic)
    {
      x [0] = -V[0]-U[0];
      x [1] = -V[1]-U[1];
      x [2] = -V[2]-U[2];
    }
    else
    {
      x [0] = -U[0];
      x [1] = -U[1];
      x [2] = -U[2];
    }

    NNCOPY (W, T);
  }
  break;
  case _FIXDIR_:
  {
    x [0] = -R[0];
    x [1] = -R[1];
    if (dynamic) x [2] = -V[2]-U[2];
    else x [2] = -U[2];

    T [1] = T [3] = T [6] = T [7] = 0.0;
    T [0] = T [4] = 1.0;
    T [2] = W [2];
    T [5] = W [5];
    T [8] = W [8];
  }
  break;
  case _VELODIR_:
  {
    x [0] = -R[0];
    x [1] = -R[1];
    x [2] = mat[0]-U[2];

    T [1] = T [3] = T [6] = T [7] = 0.0;
    T [0] = T [4] = 1.0;
    T [2] = W [2];
    T [5] = W [5];
    T [8] = W [8];
  }
  break;
  case _RIGLNK_:
  {
    float h = step * (dynamic ? 0.5 : 1.0),
	  d = mat [0],
	  delta;

    x [0] = -R[0];
    x [1] = -R[1];
    delta = d*d - h*h*DOT2(U,U);
    if (delta >= 0.0) x [2] = (sqrt (delta) - d)/h - U[2];
    else x [2] = -U[2];

    T [1] = T [3] = T [6] = T [7] = 0.0;
    T [0] = T [4] = 1.0;
    T [2] = W [2];
    T [5] = W [5];
    T [8] = W [8];
  }
  break;
  case _CONTACT_:
  {
    float X [9], Y [9], dF [9], S [3], F [3], m [3],
	  fri = mat [0],
	  res = mat [1],
	  coh = mat [2],
	  gap = mat [3];

    float udash, ulen, sdot, slen, l1, l2, u1[3], u2[3], eps2,
	 fri2, onefri2, sq1, sq2, g1, g2, dg1, dg2, a, b, c, d;

    eps2 = eps*eps;
    fri2 = fri*fri;
    onefri2 = 1.0 + fri2;
    if (dynamic) udash = (U[2] + res * MIN (V[2], 0));
    else udash = ((MAX(gap, 0)/step) + U[2]);
    ulen = sqrt (DOT2(U, U) + eps2);

    F [0] = U[0];
    F [1] = U[1];
    F [2] = (udash + fri * (ulen - eps));

    SUB (R, F, S);
    S [2] += coh;

    sdot = DOT2 (S, S);
    slen = sqrt (sdot);
    l1 = -(S[2] + fri*slen) / onefri2;
    l2 =  (slen - fri*S[2]) / onefri2;
    if (slen != 0.0)
    {
      u2[0] = S[0]/slen;
      u2[1] = S[1]/slen;
      u2[2] = -fri;
      u1[0] = -fri*u2[0];
      u1[1] = -fri*u2[1];
      u1[2] = -1.0;
    }
    else
    {
      u2[0] =  1.0;
      u2[1] =  0.0;
      u2[2] = -fri;
      u1[0] = -fri;
      u1[1] =  0.0;
      u1[2] = -1.0;
    }
    sq1 = sqrt (l1*l1 + 4.0*eps2);
    sq2 = sqrt (l2*l2 + 4.0*eps2*fri2);
    g1 = 0.5 * (sq1 + l1);
    g2 = 0.5 * (sq2 + l2);

    m [0] = g1*u1[0] + g2*u2[0];
    m [1] = g1*u1[1] + g2*u2[1];
    m [2] = g1*u1[2] + g2*u2[2];

    ADD (F, m, x);
    SCALE (x, -1.0);

    dF [1] = dF [3] = dF [6] = dF [7] = 0.0;
    dF [0] = dF [4] = dF [8] = 1.0;
    dF [2] = fri * U[0] / ulen;
    dF [5] = fri * U[1] / ulen;

    dg1 = 0.5*(1.0+l1/sq1);
    dg2 = 0.5*(1.0+l2/sq2);
    a = 0.5*(1.0+(l2 + fri*l1)/(sq2 + fri*sq1));
    b = (fri2 * dg1 + dg2) / onefri2;
    c = (fri * (dg1 - dg2)) / onefri2;
    d = (dg1 + fri2 * dg2) / onefri2;

    if (slen != 0.0)
    {
      Y [0] = a + (b - a) * S[0]*S[0] / sdot;
      Y [1] = (b - a) * S[1]*S[0] / sdot;
      Y [2] = c * S[0] / slen;
      Y [3] = Y[1];
      Y [4] = a + (b - a) * S[1]*S[1] / sdot;
      Y [5] = c * S[1] / slen;
      Y [6] = Y[2];
      Y [7] = Y[5];
      Y [8] = d;
    }
    else
    {
      Y[1] = Y[2] = Y[3] = Y[5] = Y[6] = Y[7] = 0.0;
      Y[0] = Y[4] = Y[8] = dg1;
    }

    NNMUL (Y, dF, X);
    NNSUB (dF, X, X); /* X = [I - dm/dS] dF/dU */

    NNMUL (X, W, T);
    NNADD (T, Y, T);
  }
  break;
  }

  /* 3x3 Gauss elimination */
  T [3] /= T[0]; T [6] /= T[0]; x [0] /= T[0];
  T [4] -= T[3]*T[1]; T [7] -= T[6]*T[1]; x [1] -= x[0]*T[1];
  T [5] -= T[3]*T[2]; T [8] -= T[6]*T[2]; x [2] -= x[0]*T[2];
  T [7] /= T [4]; x [1] /= T[4];
  T [8] -= T[7]*T[5]; x [2] -= x[1]*T[5];
  x [2] /= T [8];
  x [1] = x[1] - T[7]*x[2];
  x [0] = x[0] - T[3]*x[1] - T[6]*x[2];

  DR [0] = gamma * DR[0] + theta * x[0];
  DR [1] = gamma * DR[1] + theta * x[1];
  DR [2] = gamma * DR[2] + theta * x[2];
  ACC (DR, R);

  if (kind [con] == _CONTACT_) /* projection */
  {
    float fri = mat [0],
	  coh = mat [2],
          slen, l1, l2, u1[3], u2[3], g1, g2, fri2, m [3], S [3];

    COPY (R, S);
    S [2] += coh;
    fri2 = fri*fri;
    slen = LEN2 (S);
    l1 = -(S[2] + fri*slen) / (1.0 + fri2);
    l2 =  (slen - fri*S[2]) / (1.0 + fri2);
    if (slen != 0.0)
    {
      u2[0] = S[0]/slen;
      u2[1] = S[1]/slen;
      u2[2] = -fri;
      u1[0] = -fri*u2[0];
      u1[1] = -fri*u2[1];
      u1[2] = -1.0;
    }
    else
    {
      u2[0] =  1.0;
      u2[1] =  0.0;
      u2[2] = -fri;
      u1[0] = -fri;
      u1[1] =  0.0;
      u1[2] = -1.0;
    }
    g1 = MAX (l1, 0.0);
    g2 = MAX (l2, 0.0);
    m [0] = g1*u1[0] + g2*u2[0];
    m [1] = g1*u1[1] + g2*u2[1];
    m [2] = g1*u1[2] + g2*u2[2];
    S [2] -= coh;
    SUB (S, m, R);
  }
}

/* merit function */
__global__ void merit (const int dynamic, const float step, const int num, const int *kind,
  const float *mat0, const float *A0, const float *V0, const float *U0, const float *R0, float *value)
{
  int con = blockDim.x * blockIdx.x + threadIdx.x;

  /* TODO */
}

extern "C"
{
#include "../err.h"
#include "../mem.h"
#include "../dom.h"
#include "../ldy.h"
#include "../map.h"
#include "../tmr.h"
#include "../lis.h"

#define ASSERT_CUDA(call) \
if((call) != hipSuccess)\
{\
  hipError_t err = hipGetLastError(); \
  cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
  THROW (ERR_CUDA);\
}

/* LOCDYN diagobal block list sorting */
#define DIABLE(i, j) ((i)->con->kind <= (j)->con->kind)
IMPLEMENT_LIST_SORT (DOUBLY_LINKED, locdyn_sort, DIAB, p, n, DIABLE)

/* PQN solver; returns the number of iterations and writes the merit function history */
int CUDA_PQN_Solve (LOCDYN *ldy, float meritval, int maxiter, float theta, float epsilon, double *merhist)
{
  float *d_R,
        *d_U,
        *d_V,
        *d_B,
        *d_A,
        *d_W,
        *d_DR,
        *d_mat;

  int *d_kind,
      *d_ptr,
      *d_adj;

  int size, num, n, *imem;
  float *fmem;
  void *mem;
  DIAB *dia;
  OFFB *blk;
  CON *con;

  /* sort constraints by kind */
  ldy->dia = locdyn_sort (ldy->dia);

  /* number constraints */
  for (dia = ldy->dia, size = num = 0; dia; dia = dia->n, num ++)
  {
    con = dia->con;
    con->num = num;
    size += dia->nadj + 1;
  }

  /* allocate device memory */
  ASSERT_CUDA (hipMalloc((void**)&d_R, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_U, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_V, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_B, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_A, num * sizeof(float [9])));
  ASSERT_CUDA (hipMalloc((void**)&d_W, size * sizeof(float [9])));
  ASSERT_CUDA (hipMalloc((void**)&d_DR, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_mat, num * sizeof(float [4])));
  ASSERT_CUDA (hipMalloc((void**)&d_kind, num * sizeof(int)));
  ASSERT_CUDA (hipMalloc((void**)&d_ptr, (num+1) * sizeof(int)));
  ASSERT_CUDA (hipMalloc((void**)&d_adj, (size) * sizeof(int)));

  /* allocate host memory */
  ERRMEM (mem = malloc (size * sizeof (float [9])));

  /* copy R */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *R = dia->R; COPY (R, fmem); }
  ASSERT_CUDA (hipMemcpy(d_R, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy U */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *U = dia->U; COPY (U, fmem); }
  ASSERT_CUDA (hipMemcpy(d_U, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy V */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *V = dia->V; COPY (V, fmem); }
  ASSERT_CUDA (hipMemcpy(d_V, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy B */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *B = dia->B; COPY (B, fmem); }
  ASSERT_CUDA (hipMemcpy(d_B, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy A */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 9) { double *A = dia->A; NNCOPY (A, fmem); }
  ASSERT_CUDA (hipMemcpy(d_A, mem, num * sizeof(float [9]), hipMemcpyHostToDevice));
  /* copy W */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n)
  {
    double *W = dia->W;
    NNCOPY (W, fmem);
    for (blk = dia->adj, fmem += 9; blk; blk = blk->n, fmem += 9)
    {
      W = blk->W;
      NNCOPY (W, fmem); 
    }
  }
  ASSERT_CUDA (hipMemcpy(d_W, mem, size * sizeof(float [9]), hipMemcpyHostToDevice));
  /* zero DR */
  ASSERT_CUDA (hipMemset(d_DR, 0, num * sizeof(float [3])));
  /* copy mat */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 4)
  {
    con = dia->con;
    switch (con->kind)
    {
    case _VELODIR_:
      fmem [0] = VELODIR(con->Z);
      break;
    case _RIGLNK_:
      fmem [0] = RIGLNK_LEN(con->Z);
      break;
    case _CONTACT_:
      fmem [0] = con->mat.base->friction;
      fmem [1] = con->mat.base->restitution;
      fmem [2] = SURFACE_MATERIAL_Cohesion_Get (&con->mat) * con->area;
      fmem [3] = con->gap;
      break;
    default:
      break;
    }
  }
  ASSERT_CUDA (hipMemcpy(d_mat, mem, num * sizeof(float [4]), hipMemcpyHostToDevice));
  /* copy kind */
  for (dia = ldy->dia, imem = (int*) mem; dia; dia = dia->n, imem ++) { imem [0] = dia->con->kind; }
  ASSERT_CUDA (hipMemcpy(d_kind, mem, num * sizeof(int), hipMemcpyHostToDevice));
  /* copy ptr */
  for (dia = ldy->dia, imem = (int*) mem, imem [0] = 0; dia; dia = dia->n, imem ++) { imem [1] = dia->nadj + 1; }
  for (n = 0, imem = (int*) mem; n < num; n ++) imem [n+1] += imem [n];
  ASSERT_CUDA (hipMemcpy(d_ptr, mem, (num+1) * sizeof(int), hipMemcpyHostToDevice));
  /* copy adj */
  for (dia = ldy->dia, imem = (int*) mem; dia; dia = dia->n)
  {
    imem [0] = dia->con->num;
    for (blk = dia->adj, imem ++; blk; blk = blk->n, imem ++)
      imem [0] = blk->dia->con->num;
  }
  ASSERT_CUDA (hipMemcpy(d_adj, mem, size * sizeof(int), hipMemcpyHostToDevice));


  /* TODO */

  return 0;
}
}
