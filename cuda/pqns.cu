#include "hip/hip_runtime.h"
/*
 * pqns.cu
 * Copyright (C) 2010, Tomasz Koziara (t.koziara AT gmail.com)
 * --------------------------------------------------------------
 * projected quasi-Newton solver
 */

/* This file is part of Solfec.
 * Solfec is free software: you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your
 * option) any later version.
 *
 * Solfec is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public
 * License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with Solfec. If not, see <http://www.gnu.org/licenses/>. */

#include <iostream>
#include "alg.h"
using std::cerr;
using std::endl;

/* constraint kinds */
enum {_CONTACT_ = 0, _FIXPNT_, _FIXDIR_, _VELODIR_, _RIGLNK_, _GLUE_};

/* R texture sampler */
texture <float, 1, hipReadModeElementType> texR;

#define VECTOR_U_WR_B 1  /* vector version of U = WR + B */

#if VECTOR_U_WR_B /* vector version of U = WR + B */
/* vector update local velocity U = WR + B */
__global__ void U_WR_B (const int num, const int *ptr, const int *adj,
  const float *W0, const float *R0, const float *B0, float *U0)
{
  __shared__ float vals [1024];
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x ; /* global thread index */
  int warp_id = thread_id / 32; /* global warp index */
  int lane = thread_id & (32 - 1); /* thread index within the warp */
  int con = warp_id;  /* one warp per row */

  if (con < num)
  {
    const float *W;
    float *WR = &vals [threadIdx.x*3];
    int j, j0 = ptr [con], j1 = ptr [con+1];

    /* compute running sum per thread */
    SET (WR, 0.0);
    for (j = j0 + lane, W = &W0[j*9]; j < j1; j += 32, W += 288) /* 32*9 = 288 */
    {
      const int nei = adj[j]*3;
#if 0
      const float *R = &R0[nei];
#else
      const float R [3] = { tex1Dfetch (texR, nei),
                            tex1Dfetch (texR, nei+1),
                            tex1Dfetch (texR, nei+2) };
#endif
      NVADDMUL (WR, W, R, WR);
    }
  
    /* parallel reduction in shared memory */
    if (lane < 16)
    {
      float *WR1 = &vals [(threadIdx.x + 16)*3];
      ACC (WR1, WR);
    }
    if (lane < 8)
    {
      float *WR1 = &vals [(threadIdx.x + 8)*3];
      ACC (WR1, WR);
    }
    if (lane < 4)
    {
      float *WR1 = &vals [(threadIdx.x + 4)*3];
      ACC (WR1, WR);
    }
    if (lane < 2)
    {
      float *WR1 = &vals [(threadIdx.x + 2)*3];
      ACC (WR1, WR);
    }
    if (lane < 1)
    {
      float *WR1 = &vals [(threadIdx.x + 1)*3];
      ACC (WR1, WR);
    }

    /* first thread writes the result */
    if (lane == 0)
    {
      const float *B = &B0[con*3];
      float *U = &U0[con*3];
      ADD (WR, B, U);
    }
  }
}
#else
/* scalar update local velocity U = WR + B */
__global__ void U_WR_B (const int num, const int *ptr, const int *adj,
  const float *W0, const float *R0, const float *B0, float *U0)
{
  int con = blockDim.x * blockIdx.x + threadIdx.x;

  if (con < num)
  {
    const float *B = &B0[con*3];
    float WR[3] = {B[0], B[1], B[2]};
    int j, j0 = ptr [con], j1 = ptr [con+1];
    const float *W = &W0[j0*9];
    for (j = j0; j < j1; j ++, W += 9)
    { 
      const int nei = adj[j]*3;
#if 0
      const float *R = &R0[nei];
#else
      const float R [3] = { tex1Dfetch (texR, nei),
                            tex1Dfetch (texR, nei+1),
                            tex1Dfetch (texR, nei+2) };
#endif
      NVADDMUL (WR, W, R, WR);
    }
    float *U = &U0[con*3];
    COPY (WR, U);
  }
}
#endif

/* compute reaction increments and per-constraint merit function numerators */
__global__ void increments_and_merits (const int dynamic, const float step, const float theta, const float eps,
  const int num, const int *kind, const float *mat0, const int *ptr, const float *W0, const float *A0,
  const float *V0, const float *U0, const float *R0, float *DR0, float *mer)
{
  int con = blockDim.x * blockIdx.x + threadIdx.x;

  if (con < num)
  {
    const float *U = &U0[con*3],
                *R = &R0[con*3],
		*V = &V0[con*3],
		*A = &A0[con*9],
		*mat = &mat0[con*4],
		*W = &W0[ptr[con]*9];

    float *DR = &DR0[con*3];

    float x [3], T [9], up, y [3], gamma = 1.0 - theta;

    switch (kind [con])
    {
    case _FIXPNT_:
    case _GLUE_:
    {
      if (dynamic)
      {
	x [0] = -V[0]-U[0];
	x [1] = -V[1]-U[1];
	x [2] = -V[2]-U[2];
      }
      else
      {
	x [0] = -U[0];
	x [1] = -U[1];
	x [2] = -U[2];
      }

      NNCOPY (W, T);
      NVMUL (A, x, y);
      up = DOT (y, x);
    }
    break;
    case _FIXDIR_:
    {
      x [0] = -R[0];
      x [1] = -R[1];
      if (dynamic) x [2] = -V[2]-U[2];
      else x [2] = -U[2];

      T [1] = T [3] = T [6] = T [7] = 0.0;
      T [0] = T [4] = 1.0;
      T [2] = W [2];
      T [5] = W [5];
      T [8] = W [8];

      up = A[8] * x[2] * x[2];
    }
    break;
    case _VELODIR_:
    {
      x [0] = -R[0];
      x [1] = -R[1];
      x [2] = mat[0]-U[2];

      T [1] = T [3] = T [6] = T [7] = 0.0;
      T [0] = T [4] = 1.0;
      T [2] = W [2];
      T [5] = W [5];
      T [8] = W [8];

      up = A[8] * x[2] * x[2];
    }
    break;
    case _RIGLNK_:
    {
      float h = step * (dynamic ? 0.5 : 1.0),
	    d = mat [0],
	    delta;

      x [0] = -R[0];
      x [1] = -R[1];
      delta = d*d - h*h*DOT2(U,U);
      if (delta >= 0.0) x [2] = (sqrt (delta) - d)/h - U[2];
      else x [2] = -U[2];

      T [1] = T [3] = T [6] = T [7] = 0.0;
      T [0] = T [4] = 1.0;
      T [2] = W [2];
      T [5] = W [5];
      T [8] = W [8];

      up = A[8] * x[2] * x[2];
    }
    break;
    case _CONTACT_:
    {
      float X [9], Y [9], dF [9], S [3], F [3], m [3],
	    fri = mat [0],
	    res = mat [1],
	    coh = mat [2],
	    gap = mat [3];

      float udash, ulen, sdot, slen, l1, l2, u1[3], u2[3], eps2,
	   fri2, onefri2, sq1, sq2, g1, g2, dg1, dg2, a, b, c, d;

      eps2 = eps*eps;
      fri2 = fri*fri;
      onefri2 = 1.0 + fri2;
      if (dynamic) udash = (U[2] + res * MIN (V[2], 0));
      else udash = ((MAX(gap, 0)/step) + U[2]);
      ulen = sqrt (DOT2(U, U) + eps2);

      F [0] = U[0];
      F [1] = U[1];
      F [2] = (udash + fri * (ulen - eps));

      SUB (R, F, S);
      S [2] += coh;

      sdot = DOT2 (S, S);
      slen = sqrt (sdot);
      l1 = -(S[2] + fri*slen) / onefri2;
      l2 =  (slen - fri*S[2]) / onefri2;
      if (slen != 0.0)
      {
	u2[0] = S[0]/slen;
	u2[1] = S[1]/slen;
	u2[2] = -fri;
	u1[0] = -fri*u2[0];
	u1[1] = -fri*u2[1];
	u1[2] = -1.0;
      }
      else
      {
	u2[0] =  1.0;
	u2[1] =  0.0;
	u2[2] = -fri;
	u1[0] = -fri;
	u1[1] =  0.0;
	u1[2] = -1.0;
      }
      sq1 = sqrt (l1*l1 + 4.0*eps2);
      sq2 = sqrt (l2*l2 + 4.0*eps2*fri2);
      g1 = 0.5 * (sq1 + l1);
      g2 = 0.5 * (sq2 + l2);

      m [0] = g1*u1[0] + g2*u2[0];
      m [1] = g1*u1[1] + g2*u2[1];
      m [2] = g1*u1[2] + g2*u2[2];

      ADD (F, m, x);
      SCALE (x, -1.0);
      NVMUL (A, x, y);
      up = DOT (y, x);

      dF [1] = dF [3] = dF [6] = dF [7] = 0.0;
      dF [0] = dF [4] = dF [8] = 1.0;
      dF [2] = fri * U[0] / ulen;
      dF [5] = fri * U[1] / ulen;

      dg1 = 0.5*(1.0+l1/sq1);
      dg2 = 0.5*(1.0+l2/sq2);
      a = 0.5*(1.0+(l2 + fri*l1)/(sq2 + fri*sq1));
      b = (fri2 * dg1 + dg2) / onefri2;
      c = (fri * (dg1 - dg2)) / onefri2;
      d = (dg1 + fri2 * dg2) / onefri2;

      if (slen != 0.0)
      {
	Y [0] = a + (b - a) * S[0]*S[0] / sdot;
	Y [1] = (b - a) * S[1]*S[0] / sdot;
	Y [2] = c * S[0] / slen;
	Y [3] = Y[1];
	Y [4] = a + (b - a) * S[1]*S[1] / sdot;
	Y [5] = c * S[1] / slen;
	Y [6] = Y[2];
	Y [7] = Y[5];
	Y [8] = d;
      }
      else
      {
	Y[1] = Y[2] = Y[3] = Y[5] = Y[6] = Y[7] = 0.0;
	Y[0] = Y[4] = Y[8] = dg1;
      }

      NNMUL (Y, dF, X);
      NNSUB (dF, X, X); /* X = [I - dm/dS] dF/dU */

      NNMUL (X, W, T);
      NNADD (T, Y, T);
    }
    break;
    }

    /* 3x3 Gauss elimination */
    T [3] /= T[0]; T [6] /= T[0]; x [0] /= T[0];
    T [4] -= T[3]*T[1]; T [7] -= T[6]*T[1]; x [1] -= x[0]*T[1];
    T [5] -= T[3]*T[2]; T [8] -= T[6]*T[2]; x [2] -= x[0]*T[2];
    T [7] /= T [4]; x [1] /= T[4];
    T [8] -= T[7]*T[5]; x [2] -= x[1]*T[5];
    x [2] /= T [8];
    x [1] = x[1] - T[7]*x[2];
    x [0] = x[0] - T[3]*x[1] - T[6]*x[2];

    /* increment */
    DR [0] = gamma * DR[0] + theta * x[0];
    DR [1] = gamma * DR[1] + theta * x[1];
    DR [2] = gamma * DR[2] + theta * x[2];

    /* merit */
    mer [con] = up;
  }
}

#define ACCUM_N 1024
/* reduce merit function */
__global__ void reduce_merit (const int num, const float *mer, float *out)
{
  __shared__ float accum [ACCUM_N];

  for (int i = threadIdx.x; i < ACCUM_N; i += blockDim.x) /* for each accumulation index */
  {
      float sum = 0;

      for (int j = i; j < num; j += ACCUM_N) sum += mer [j]; /* sum up every ACCUM_N(th) entry */

      accum [i] = sum;
  }

  for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1) /* tree-like reduction */
  {
    __syncthreads();

    for (int i = threadIdx.x; i < stride; i += blockDim.x) accum [i] += accum [stride + i];
  }

  if (threadIdx.x == 0) out [0] = accum [0];
}

/* increment reactions */
__global__ void increment_reactions (const int num, const int *kind, const float *mat0, float *DR0, float *R0)
{
  int con = blockDim.x * blockIdx.x + threadIdx.x;

  if (con < num)
  {
    const float *mat = &mat0[con*4];

    float *R = &R0[con*3],
	  *DR = &DR0[con*3];

    ACC (DR, R); /* increment reaction */

    if (kind [con] == _CONTACT_) /* projection */
    {
      float fri = mat [0],
	    coh = mat [2],
	    slen, l1, l2, u1[3], u2[3], g1, g2, fri2, m [3], S [3];

      COPY (R, S);
      S [2] += coh;
      fri2 = fri*fri;
      slen = LEN2 (S);
      l1 = -(S[2] + fri*slen) / (1.0 + fri2);
      l2 =  (slen - fri*S[2]) / (1.0 + fri2);
      if (slen != 0.0)
      {
	u2[0] = S[0]/slen;
	u2[1] = S[1]/slen;
	u2[2] = -fri;
	u1[0] = -fri*u2[0];
	u1[1] = -fri*u2[1];
	u1[2] = -1.0;
      }
      else
      {
	u2[0] =  1.0;
	u2[1] =  0.0;
	u2[2] = -fri;
	u1[0] = -fri;
	u1[1] =  0.0;
	u1[2] = -1.0;
      }
      g1 = MAX (l1, 0.0);
      g2 = MAX (l2, 0.0);
      m [0] = g1*u1[0] + g2*u2[0];
      m [1] = g1*u1[1] + g2*u2[1];
      m [2] = g1*u1[2] + g2*u2[2];
      S [2] -= coh;
      SUB (S, m, R);
    }
  }
}

extern "C"
{

#include "../err.h"
#include "../mem.h"
#include "../dom.h"
#include "../ldy.h"
#include "../map.h"
#include "../tmr.h"
#include "../lis.h"

#define ASSERT_CUDA(call) \
if((call) != hipSuccess)\
{\
  hipError_t err = hipGetLastError(); \
  cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
  THROW (ERR_CUDA);\
}

/* LOCDYN diagobal block list sorting */
#define DIABLE(i, j) ((i)->con->kind <= (j)->con->kind)
IMPLEMENT_LIST_SORT (DOUBLY_LINKED, diab_sort, DIAB, p, n, DIABLE)

#if VECTOR_U_WR_B
/* LOCDYN off-diagobal block list sorting */
#define OFFBLE(i, j) ((i)->dia->con->num <= (j)->dia->con->num)
IMPLEMENT_LIST_SORT (SINGLE_LINKED, offb_sort, OFFB, p, n, OFFBLE)
#endif

/* PQN solver; returns the number of iterations and writes the merit function history */
int CUDA_PQN_Solve (LOCDYN *ldy, double meritval, int maxiter, double theta, double epsilon, double *merhist)
{
  float *d_R,
        *d_R0,
        *d_U,
        *d_V,
        *d_B,
        *d_A,
        *d_W,
        *d_DR,
        *d_mat,
        *d_mer,
        *d_out,
         h_out;

  int *d_kind,
      *d_ptr,
      *d_adj;

  int size, num, n, *imem;
  float *fmem;
  void *mem;
  DIAB *dia;
  OFFB *blk;
  CON *con;

  /* sort constraints by kind */
  ldy->dia = diab_sort (ldy->dia);

  /* number constraints */
  for (dia = ldy->dia, size = num = 0; dia; dia = dia->n, num ++)
  {
    con = dia->con;
    con->num = num;
    size += dia->nadj + 1;
  }

#if VECTOR_U_WR_B
  /* sort off-diagonal blocks */
  for (dia = ldy->dia; dia; dia = dia->n)
  {
    dia->adj = offb_sort (dia->adj);
  }
#endif

  /* allocate device memory */
  ASSERT_CUDA (hipMalloc((void**)&d_R, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_R0, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_U, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_V, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_B, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_A, num * sizeof(float [9])));
  ASSERT_CUDA (hipMalloc((void**)&d_W, size * sizeof(float [9])));
  ASSERT_CUDA (hipMalloc((void**)&d_DR, num * sizeof(float [3])));
  ASSERT_CUDA (hipMalloc((void**)&d_mat, num * sizeof(float [4])));
  ASSERT_CUDA (hipMalloc((void**)&d_mer, num * sizeof(float)));
  ASSERT_CUDA (hipMalloc((void**)&d_out, sizeof(float)));
  ASSERT_CUDA (hipMalloc((void**)&d_kind, num * sizeof(int)));
  ASSERT_CUDA (hipMalloc((void**)&d_ptr, (num+1) * sizeof(int)));
  ASSERT_CUDA (hipMalloc((void**)&d_adj, size * sizeof(int)));

  /* allocate host memory */
  ERRMEM (mem = malloc (size * sizeof (float [9])));

  /* copy R, R0 */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *R = dia->R; COPY (R, fmem); }
  ASSERT_CUDA (hipMemcpy(d_R, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  if (num) ASSERT_CUDA (hipBindTexture (0, texR, d_R, num * sizeof (float [3])));
  ASSERT_CUDA (hipMemcpy(d_R0, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy U */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *U = dia->U; COPY (U, fmem); }
  ASSERT_CUDA (hipMemcpy(d_U, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy V */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *V = dia->V; COPY (V, fmem); }
  ASSERT_CUDA (hipMemcpy(d_V, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy B */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *B = dia->B; COPY (B, fmem); }
  ASSERT_CUDA (hipMemcpy(d_B, mem, num * sizeof(float [3]), hipMemcpyHostToDevice));
  /* copy A */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 9) { double *A = dia->A; NNCOPY (A, fmem); }
  ASSERT_CUDA (hipMemcpy(d_A, mem, num * sizeof(float [9]), hipMemcpyHostToDevice));
  /* copy W */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n)
  {
    double *W = dia->W;
    NNCOPY (W, fmem);
    for (blk = dia->adj, fmem += 9; blk; blk = blk->n, fmem += 9)
    {
      W = blk->W;
      NNCOPY (W, fmem); 
    }
  }
  ASSERT_CUDA (hipMemcpy(d_W, mem, size * sizeof(float [9]), hipMemcpyHostToDevice));
  /* zero DR */
  ASSERT_CUDA (hipMemset(d_DR, 0, num * sizeof(float [3])));
  /* copy mat */
  for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 4)
  {
    con = dia->con;
    switch (con->kind)
    {
    case _VELODIR_:
      fmem [0] = VELODIR(con->Z);
      break;
    case _RIGLNK_:
      fmem [0] = RIGLNK_LEN(con->Z);
      break;
    case _CONTACT_:
      fmem [0] = con->mat.base->friction;
      fmem [1] = con->mat.base->restitution;
      fmem [2] = SURFACE_MATERIAL_Cohesion_Get (&con->mat) * con->area;
      fmem [3] = con->gap;
      break;
    default:
      break;
    }
  }
  ASSERT_CUDA (hipMemcpy(d_mat, mem, num * sizeof(float [4]), hipMemcpyHostToDevice));
  /* copy kind */
  for (dia = ldy->dia, imem = (int*) mem; dia; dia = dia->n, imem ++) { imem [0] = dia->con->kind; }
  ASSERT_CUDA (hipMemcpy(d_kind, mem, num * sizeof(int), hipMemcpyHostToDevice));
  /* copy ptr */
  for (dia = ldy->dia, imem = (int*) mem, imem [0] = 0; dia; dia = dia->n, imem ++) { imem [1] = dia->nadj + 1; }
  for (n = 0, imem = (int*) mem; n < num; n ++) imem [n+1] += imem [n];
  ASSERT_CUDA (hipMemcpy(d_ptr, mem, (num+1) * sizeof(int), hipMemcpyHostToDevice));
  /* copy adj */
  for (dia = ldy->dia, imem = (int*) mem; dia; dia = dia->n)
  {
    imem [0] = dia->con->num;
    for (blk = dia->adj, imem ++; blk; blk = blk->n, imem ++)
      imem [0] = blk->dia->con->num;
  }
  ASSERT_CUDA (hipMemcpy(d_adj, mem, size * sizeof(int), hipMemcpyHostToDevice));

  /* --- solution loop --- */

  int tpb = 256;
  int bpg = (num + tpb - 1) / tpb;
  double *merit, prevm, step, merit0, mden;
  int dynamic, div, gt, iters;
  char fmt [512];

  sprintf (fmt, "NEWTON_SOLVER: theta: %%6g iteration: %%%dd merit: %%.2e\n", (int)log10 (maxiter) + 1);
  mden = ldy->free_energy > 0.0 ? ldy->free_energy : 1.0;
  dynamic = ldy->dom->dynamic;
  merit = &ldy->dom->merit;
  step = ldy->dom->step;
  *merit = meritval+1.0;
  iters = 0;
  div = 1;
  gt = 0;

  while (iters < maxiter && *merit > meritval)
  {
#if VECTOR_U_WR_B
    int vbpg = (32*num + tpb - 1) / tpb;
    U_WR_B <<<vbpg, tpb>>> (num, d_ptr, d_adj, d_W, d_R, d_B, d_U);
#else
    U_WR_B <<<bpg, tpb>>> (num, d_ptr, d_adj, d_W, d_R, d_B, d_U);
#endif

    increments_and_merits <<<bpg, tpb>>> (dynamic, step, theta, epsilon, num, d_kind, d_mat, d_ptr, d_W, d_A, d_V, d_U, d_R, d_DR, d_mer);

    reduce_merit <<<1, tpb>>> (num, d_mer, d_out);
    
    ASSERT_CUDA (hipMemcpy (&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost));

    *merit = (double) h_out / mden;

    prevm = *merit;

    merhist [iters] = *merit;

    if (iters == 0) merit0 = *merit;

    if (*merit < meritval) break;

    increment_reactions <<<bpg, tpb>>> (num, d_kind, d_mat, d_DR, d_R);

    if (*merit > prevm && ++gt > 10 && *merit > 10)
    {
      if (theta < 0.0009765625) theta = 0.5; /* < 0.5**10 */
      else theta *= 0.5;
      gt = 0;

      ASSERT_CUDA (hipMemcpy(d_R, d_R0, num * sizeof(float [3]), hipMemcpyDeviceToDevice)); /* R = R0 */
      ASSERT_CUDA (hipMemset(d_DR, 0, num * sizeof(float [3]))); /* DR = 0 */
    }

    if (ldy->dom->verbose && iters % div == 0) printf (fmt, theta, iters, *merit), div *= 2;

    iters ++;
  }

  if (ldy->dom->verbose) printf (fmt, theta, iters, *merit);

  if (*merit > merit0)
  {
    *merit = merhist [0];

    if (ldy->dom->verbose) printf ("NEWTON_SOLVER: DIVERGED => Reusing previous solution (merit: %.2e)\n", *merit);
  }
  else /* copy R, U, con->merit from GPU to CPU */
  {
    ASSERT_CUDA (hipMemcpy(mem, d_R, num * sizeof(float [3]), hipMemcpyDeviceToHost));
    for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *R = dia->R; COPY (fmem, R); }
    ASSERT_CUDA (hipMemcpy(mem, d_U, num * sizeof(float [3]), hipMemcpyDeviceToHost));
    for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem += 3) { double *U = dia->U; COPY (fmem, U); }
    ASSERT_CUDA (hipMemcpy(mem, d_mer, num * sizeof(float), hipMemcpyDeviceToHost));
    for (dia = ldy->dia, fmem = (float*) mem; dia; dia = dia->n, fmem ++) { con = dia->con; con->merit = fmem [0] / mden; }
  }

  /* free memory */
  ASSERT_CUDA (hipFree(d_R));
  if (num) ASSERT_CUDA (hipUnbindTexture (texR));
  ASSERT_CUDA (hipFree(d_R0));
  ASSERT_CUDA (hipFree(d_U));
  ASSERT_CUDA (hipFree(d_V));
  ASSERT_CUDA (hipFree(d_B));
  ASSERT_CUDA (hipFree(d_A));
  ASSERT_CUDA (hipFree(d_W));
  ASSERT_CUDA (hipFree(d_DR));
  ASSERT_CUDA (hipFree(d_mat));
  ASSERT_CUDA (hipFree(d_mer));
  ASSERT_CUDA (hipFree(d_out));
  ASSERT_CUDA (hipFree(d_kind));
  ASSERT_CUDA (hipFree(d_ptr));
  ASSERT_CUDA (hipFree(d_adj));
  free (mem);

  return iters;
}

} /* extern C */
