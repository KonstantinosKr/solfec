#include "hip/hip_runtime.h"
/*
 * spmv.cu
 * Copyright (C) 2010, Tomasz Koziara (t.koziara AT gmail.com)
 * --------------------------------------------------------------
 * sparse matrix-vector product
 */

/* This file is part of Solfec.
 * Solfec is free software: you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your
 * option) any later version.
 *
 * Solfec is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public
 * License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with Solfec. If not, see <http://www.gnu.org/licenses/>. */

#include <iostream>
using std::cerr;
using std::endl;

#if 1
__global__ void csrspmv (
    const int rows,
    const int *ptr,
    const float *a,
    const float *x,
    float *y)
{
  int row = blockDim.x * blockIdx.x + threadIdx.x;

  if(row < rows)
  {
    float dot = 0;
    int row_start = ptr [row];
    int row_end = ptr [row+1];
    for (int j = row_start; j < row_end; j ++) dot += a [j] * x [j];
    y [row] += dot;
  }
}
#else
__global__ void csrspmv (
    const int rows,
    const int *ptr,
    const float *a,
    const float *x,
    float *y)
{
  __shared__ float vals [256];
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x ; /* global thread index */
  int warp_id = thread_id / 32; /* global warp index */
  int lane = thread_id & (32 - 1); /* thread index within the warp */

  /* one warp per row */
  int row = warp_id ;

  if (row < rows)
  {
    int row_start = ptr [row];
    int row_end = ptr [row+1];

    /* compute running sum per thread  */
    vals [threadIdx.x] = 0;
    for (int j = row_start + lane ; j < row_end ; j += 32) vals [threadIdx.x] += a [j] * x [j];

    if (lane < 16) vals [threadIdx.x] += vals [threadIdx.x + 16];
    if (lane < 8) vals [threadIdx.x] += vals [threadIdx.x + 8];
    if (lane < 4) vals [threadIdx.x] += vals [threadIdx.x + 4];
    if (lane < 2) vals [threadIdx.x] += vals [threadIdx.x + 2];
    if (lane < 1) vals [threadIdx.x] += vals [threadIdx.x + 1];

    if (lane == 0) y [row] += vals [threadIdx.x];
  }
}
#endif

extern "C"
{
#include "../err.h"
#include "../mem.h"
#include "../ldy.h"
#include "../tmr.h"

#define ASSERT_CUDA(call) \
if((call) != hipSuccess)\
{\
  hipError_t err = hipGetLastError(); \
  cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
  THROW (ERR_CUDA);\
}

typedef struct
{
  int rows;
  int nnz;
  float *d_a;
  int *d_ptr, *h_ptr;
  float *d_x, *h_x;
  float *d_y, *h_y;
  LOCDYN *ldy;
} U_WR_B_DATA;

void* CUDA_U_WR_B_Create (LOCDYN *ldy)
{
  int i, n, m, *h_ptr;
  U_WR_B_DATA *u;
  float *h_a;
  DIAB *dia;
  OFFB *blk;

  ERRMEM (u = (U_WR_B_DATA*) MEM_CALLOC (sizeof (U_WR_B_DATA)));
  u->ldy = ldy;

  for (dia = ldy->dia, i = 0; dia; dia = dia->n, i += 3);
  u->rows = i;
 
  ERRMEM (u->h_ptr = (int*) malloc ((u->rows+1) * sizeof (int)));
  ERRMEM (u->h_y = (float*) malloc (u->rows * sizeof (float)));
  h_ptr = u->h_ptr;
  h_ptr [0] = 0;

  for (dia = ldy->dia, n = 0, i = 1; dia; dia = dia->n, i += 3)
  {
    m = (1+dia->nadj);
#if MPI
    m += dia->nadjext;
#endif
    h_ptr [i] = h_ptr [i+1] = h_ptr [i+2] = 3*m; /* nonzeros per row */
    n += m;
  }
  u->nnz = 9*n;

  ERRMEM (h_a = (float*) malloc (u->nnz * sizeof (float)));
  ERRMEM (u->h_x = (float*) malloc (u->nnz * sizeof (float)));

  for (i = 1; i <= u->rows; i ++) h_ptr [i] += h_ptr [i-1]; /* host pointers done */

  for (dia = ldy->dia, i = 0; dia; dia = dia->n, i += 3)
  {
    double *W = dia->W;
    float *a0 = &h_a  [h_ptr [i]],
          *a1 = &h_a  [h_ptr [i+1]],
          *a2 = &h_a  [h_ptr [i+2]];

    a0[0] = W[0]; a0[1] = W[3]; a0[2] = W[6];
    a1[0] = W[1]; a1[1] = W[4]; a1[2] = W[7];
    a2[0] = W[2]; a2[1] = W[5]; a2[2] = W[8];
    a0 += 3; a1 += 3; a2 += 3;
    for (blk = dia->adj; blk; blk = blk->n)
    {
      W = blk->W;
      a0[0] = W[0]; a0[1] = W[3]; a0[2] = W[6];
      a1[0] = W[1]; a1[1] = W[4]; a1[2] = W[7];
      a2[0] = W[2]; a2[1] = W[5]; a2[2] = W[8];
      a0 += 3; a1 += 3; a2 += 3;
    }
#if MPI
    for (blk = dia->adjext; blk; blk = blk->n)
    {
      W = blk->W;
      a0[0] = W[0]; a0[1] = W[3]; a0[2] = W[6];
      a1[0] = W[1]; a1[1] = W[4]; a1[2] = W[7];
      a2[0] = W[2]; a2[1] = W[5]; a2[2] = W[8];
      a0 += 3; a1 += 3; a2 += 3;
    }
#endif
  }
  
  ASSERT_CUDA (hipMalloc((void**)&u->d_a, u->nnz * sizeof(float)));
  ASSERT_CUDA (hipMalloc((void**)&u->d_x, u->nnz * sizeof(float)));
  ASSERT_CUDA (hipMalloc((void**)&u->d_y, u->rows * sizeof(float)));
  ASSERT_CUDA (hipMalloc((void**)&u->d_ptr, (u->rows+1) * sizeof(int)));

  /* copy h_a  to u->d_a */
  ASSERT_CUDA (hipMemcpy(u->d_a, h_a, u->nnz * sizeof(float), hipMemcpyHostToDevice));

  /* copy h_ptr  to u->d_ptr */
  ASSERT_CUDA (hipMemcpy(u->d_ptr, h_ptr, (u->rows+1) * sizeof(int), hipMemcpyHostToDevice));

  free (h_a);

  return u;
}

void CUDA_U_WR_B (void *U_WR_B)
{
  U_WR_B_DATA *u = (U_WR_B_DATA*) U_WR_B;
  float *h_x, *h_y;
  int *h_ptr;
  DIAB *dia;
  OFFB *blk;
  TIMING tt;
  double t[2];

  timerstart (&tt);

  /* copy reactions */
  for (dia = u->ldy->dia, h_ptr = u->h_ptr, h_x = u->h_x; dia; dia = dia->n, h_ptr += 3)
  {
    double *R = dia->R;
    float *a0 = &h_x  [h_ptr [0]],
          *a1 = &h_x  [h_ptr [1]],
          *a2 = &h_x  [h_ptr [2]];

    a0[0] = a1[0] = a2 [0] = R[0];
    a0[1] = a1[1] = a2 [1] = R[1];
    a0[2] = a1[2] = a2 [2] = R[2];
    a0 += 3; a1 += 3; a2 += 3;
    for (blk = dia->adj; blk; blk = blk->n)
    {
      R = blk->dia->R;
      a0[0] = a1[0] = a2 [0] = R[0];
      a0[1] = a1[1] = a2 [1] = R[1];
      a0[2] = a1[2] = a2 [2] = R[2];
      a0 += 3; a1 += 3; a2 += 3;
    }
#if MPI
    for (blk = dia->adjext; blk; blk = blk->n)
    {
      R = CON(blk->dia)->R;
      a0[0] = a1[0] = a2 [0] = R[0];
      a0[1] = a1[1] = a2 [1] = R[1];
      a0[2] = a1[2] = a2 [2] = R[2];
      a0 += 3; a1 += 3; a2 += 3;
    }
#endif
  }

  /* copy u->h_x  to u->d_x */
  ASSERT_CUDA (hipMemcpy(u->d_x, u->h_x, u->nnz * sizeof(float), hipMemcpyHostToDevice));

  /* zero u->d_y */
  ASSERT_CUDA (hipMemset(u->d_y, 0, u->rows * sizeof(float)));

  t[0] = timerend (&tt);
  timerstart (&tt);

  /* GPU matrix vector product */  
  int tpb = 256;
  int bpg = (u->rows + tpb - 1) / tpb;
  csrspmv <<<bpg, tpb>>> (u->rows, u->d_ptr, u->d_a, u->d_x, u->d_y);

  t[1] = timerend (&tt);
  timerstart (&tt);

  /* copy u->d_y to u->h_y */
  ASSERT_CUDA (hipMemcpy(u->h_y, u->d_y, u->rows * sizeof(float), hipMemcpyDeviceToHost));

  /* update U */
  for (dia = u->ldy->dia, h_y = u->h_y; dia; dia = dia->n, h_y += 3)
  {
    double *B = dia->B,
           *U = dia->U;

    U [0] = B[0] + h_y[0];
    U [1] = B[1] + h_y[1];
    U [2] = B[2] + h_y[2];
  }

  t[0] += timerend (&tt);
  printf ("CUDA TIMING: host %g, devi %g\n", t[0], t[1]);
}

void CUDA_U_WR_B_Destroy (void *U_WR_B)
{
  U_WR_B_DATA *u = (U_WR_B_DATA*) U_WR_B;

  ASSERT_CUDA (hipFree(u->d_a));
  ASSERT_CUDA (hipFree(u->d_x));
  ASSERT_CUDA (hipFree(u->d_y));
  ASSERT_CUDA (hipFree(u->d_ptr));
  free (u->h_ptr);
  free (u->h_x);
  free (u->h_y);
  free (u);
}
}
